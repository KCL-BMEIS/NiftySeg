#include "hip/hip_runtime.h"
/*
 *  _reg_tools_gpu.cu
 *  
 *
 *  Created by Marc Modat and Pankaj Daga on 24/03/2009.
 *  Copyright (c) 2009, University College London. All rights reserved.
 *  Centre for Medical Image Computing (CMIC)
 *  See the LICENSE.txt file in the nifty_reg root folder
 *
 */

#ifndef _REG_TOOLS_GPU_CU
#define _REG_TOOLS_GPU_CU

#include "_reg_blocksize_gpu.h"
#include "_reg_tools_kernels.cu"


void reg_voxelCentric2NodeCentric_gpu(	nifti_image *targetImage,
					nifti_image *controlPointImage,
					float4 **voxelNMIGradientArray_d,
					float4 **nodeNMIGradientArray_d,
					float weight)
{
	const int nodeNumber = controlPointImage->nx * controlPointImage->ny * controlPointImage->nz;
	const int voxelNumber = targetImage->nx * targetImage->ny * targetImage->nz;
	const int3 targetImageDim = make_int3(targetImage->nx, targetImage->ny, targetImage->nz);
	const int3 gridSize = make_int3(controlPointImage->nx, controlPointImage->ny, controlPointImage->nz);
	const float3 voxelNodeRatio_h = make_float3(
		controlPointImage->dx / targetImage->dx,
		controlPointImage->dy / targetImage->dy,
		controlPointImage->dz / targetImage->dz);

	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_NodeNumber),&nodeNumber,sizeof(int)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_TargetImageDim),&targetImageDim,sizeof(int3)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_ControlPointImageDim),&gridSize,sizeof(int3)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_VoxelNodeRatio),&voxelNodeRatio_h,sizeof(float3)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_Weight),&weight,sizeof(float)));

	CUDA_SAFE_CALL(hipBindTexture(0, gradientImageTexture, *voxelNMIGradientArray_d, voxelNumber*sizeof(float4)));

	const unsigned int Grid_reg_voxelCentric2NodeCentric = (unsigned int)ceil((float)nodeNumber/(float)Block_reg_voxelCentric2NodeCentric);
	dim3 B1(Block_reg_voxelCentric2NodeCentric,1,1);
	dim3 G1(Grid_reg_voxelCentric2NodeCentric,1,1);

	reg_voxelCentric2NodeCentric_kernel <<< G1, B1 >>> (*nodeNMIGradientArray_d);
	CUDA_SAFE_CALL(hipDeviceSynchronize());
#ifndef NDEBUG
    printf("[NiftyReg CUDA DEBUG] reg_voxelCentric2NodeCentric_gpu kernel: %s - Grid size [%i %i %i] - Block size [%i %i %i]\n",
	       hipGetErrorString(hipGetLastError()),G1.x,G1.y,G1.z,B1.x,B1.y,B1.z);
#endif
}

void reg_convertNMIGradientFromVoxelToRealSpace_gpu(	mat44 *sourceMatrix_xyz,
							nifti_image *controlPointImage,
							float4 **nodeNMIGradientArray_d)
{
	const int nodeNumber = controlPointImage->nx * controlPointImage->ny * controlPointImage->nz;
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_NodeNumber),&nodeNumber,sizeof(int)));

    float4 *matrix_h;CUDA_SAFE_CALL(hipHostMalloc(&matrix_h, 3*sizeof(float4)));
	matrix_h[0] = make_float4(sourceMatrix_xyz->m[0][0], sourceMatrix_xyz->m[0][1], sourceMatrix_xyz->m[0][2], sourceMatrix_xyz->m[0][3]);
	matrix_h[1] = make_float4(sourceMatrix_xyz->m[1][0], sourceMatrix_xyz->m[1][1], sourceMatrix_xyz->m[1][2], sourceMatrix_xyz->m[1][3]);
	matrix_h[2] = make_float4(sourceMatrix_xyz->m[2][0], sourceMatrix_xyz->m[2][1], sourceMatrix_xyz->m[2][2], sourceMatrix_xyz->m[2][3]);
	float4 *matrix_d;
    CUDA_SAFE_CALL(hipMalloc(&matrix_d, 3*sizeof(float4)));
	CUDA_SAFE_CALL(hipMemcpy(matrix_d, matrix_h, 3*sizeof(float4), hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipHostFree((void *)matrix_h));
	CUDA_SAFE_CALL(hipBindTexture(0, matrixTexture, matrix_d, 3*sizeof(float4)));
	
	const unsigned int Grid_reg_convertNMIGradientFromVoxelToRealSpace =
		(unsigned int)ceil((float)nodeNumber/(float)Block_reg_convertNMIGradientFromVoxelToRealSpace);
	dim3 B1(Grid_reg_convertNMIGradientFromVoxelToRealSpace,1,1);
	dim3 G1(Block_reg_convertNMIGradientFromVoxelToRealSpace,1,1);

	_reg_convertNMIGradientFromVoxelToRealSpace_kernel <<< G1, B1 >>> (*nodeNMIGradientArray_d);
	CUDA_SAFE_CALL(hipDeviceSynchronize());
#ifndef NDEBUG
    printf("[NiftyReg CUDA DEBUG] reg_convertNMIGradientFromVoxelToRealSpace: %s - Grid size [%i %i %i] - Block size [%i %i %i]\n",
	       hipGetErrorString(hipGetLastError()),G1.x,G1.y,G1.z,B1.x,B1.y,B1.z);
#endif
	CUDA_SAFE_CALL(hipFree(matrix_d));
}


void reg_initialiseConjugateGradient(	float4 **nodeNMIGradientArray_d,
					float4 **conjugateG_d,
					float4 **conjugateH_d,
					int nodeNumber)
{
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_NodeNumber),&nodeNumber,sizeof(int)));
	CUDA_SAFE_CALL(hipBindTexture(0, gradientImageTexture, *nodeNMIGradientArray_d, nodeNumber*sizeof(float4)));

	const unsigned int Grid_reg_initialiseConjugateGradient =
		(unsigned int)ceil((float)nodeNumber/(float)Block_reg_initialiseConjugateGradient);
	dim3 B1(Grid_reg_initialiseConjugateGradient,1,1);
	dim3 G1(Block_reg_initialiseConjugateGradient,1,1);

	reg_initialiseConjugateGradient_kernel <<< G1, B1 >>> (*conjugateG_d);
	CUDA_SAFE_CALL(hipDeviceSynchronize());
#ifndef NDEBUG
    printf("[NiftyReg CUDA DEBUG] reg_initialiseConjugateGradient: %s - Grid size [%i %i %i] - Block size [%i %i %i]\n",
	       hipGetErrorString(hipGetLastError()),G1.x,G1.y,G1.z,B1.x,B1.y,B1.z);
#endif
	CUDA_SAFE_CALL(hipMemcpy(*conjugateH_d, *conjugateG_d, nodeNumber*sizeof(float4), hipMemcpyDeviceToDevice));
}

void reg_GetConjugateGradient(	float4 **nodeNMIGradientArray_d,
				float4 **conjugateG_d,
				float4 **conjugateH_d,
				int nodeNumber)
{
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_NodeNumber),&nodeNumber,sizeof(int)));
	CUDA_SAFE_CALL(hipBindTexture(0, conjugateGTexture, *conjugateG_d, nodeNumber*sizeof(float4)));
	CUDA_SAFE_CALL(hipBindTexture(0, conjugateHTexture, *conjugateH_d, nodeNumber*sizeof(float4)));
	CUDA_SAFE_CALL(hipBindTexture(0, gradientImageTexture, *nodeNMIGradientArray_d, nodeNumber*sizeof(float4)));

	// gam = sum((grad+g)*grad)/sum(HxG);
	const unsigned int Grid_reg_GetConjugateGradient1 = (unsigned int)ceil((float)nodeNumber/(float)Block_reg_GetConjugateGradient1);
	dim3 B1(Block_reg_GetConjugateGradient1,1,1);
	dim3 G1(Grid_reg_GetConjugateGradient1,1,1);

	float2 *sum_d;
    CUDA_SAFE_CALL(hipMalloc(&sum_d, nodeNumber*sizeof(float2)));
	reg_GetConjugateGradient1_kernel <<< G1, B1 >>> (sum_d);
	CUDA_SAFE_CALL(hipDeviceSynchronize());
#ifndef NDEBUG
    printf("[NiftyReg CUDA DEBUG] reg_GetConjugateGradient1 kernel: %s - Grid size [%i %i %i] - Block size [%i %i %i]\n",
	       hipGetErrorString(hipGetLastError()),G1.x,G1.y,G1.z,B1.x,B1.y,B1.z);
#endif
    float2 *sum_h;CUDA_SAFE_CALL(hipHostMalloc(&sum_h, nodeNumber*sizeof(float2)));
	CUDA_SAFE_CALL(hipMemcpy(sum_h,sum_d, nodeNumber*sizeof(float2),hipMemcpyDeviceToHost));
	CUDA_SAFE_CALL(hipFree(sum_d));
	double dgg = 0.0;
	double gg = 0.0;
	for(int i=0; i<nodeNumber; i++){
		dgg += sum_h[i].x;
		gg += sum_h[i].y;
	}
	float gam = (float)(dgg / gg);
	CUDA_SAFE_CALL(hipHostFree((void *)sum_h));

	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_ScalingFactor),&gam,sizeof(float)));
	const unsigned int Grid_reg_GetConjugateGradient2 = (unsigned int)ceil((float)nodeNumber/(float)Block_reg_GetConjugateGradient2);
	dim3 B2(Block_reg_GetConjugateGradient2,1,1);
	dim3 G2(Grid_reg_GetConjugateGradient2,1,1);
	reg_GetConjugateGradient2_kernel <<< G2, B2 >>> (*nodeNMIGradientArray_d, *conjugateG_d, *conjugateH_d);
	CUDA_SAFE_CALL(hipDeviceSynchronize());
#ifndef NDEBUG
    printf("[NiftyReg CUDA DEBUG] reg_GetConjugateGradient2 kernel: %s - Grid size [%i %i %i] - Block size [%i %i %i]\n",
	       hipGetErrorString(hipGetLastError()),G1.x,G1.y,G1.z,B1.x,B1.y,B1.z);
#endif


}

float reg_getMaximalLength_gpu(	float4 **nodeNMIGradientArray_d,
				int nodeNumber)
{

	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_NodeNumber),&nodeNumber,sizeof(int)));
	CUDA_SAFE_CALL(hipBindTexture(0, gradientImageTexture, *nodeNMIGradientArray_d, nodeNumber*sizeof(float4)));

	// each thread extract the maximal value out of 128
	const int threadNumber = (int)ceil((float)nodeNumber/128.0f);
	const unsigned int Grid_reg_getMaximalLength = (unsigned int)ceil((float)threadNumber/(float)Block_reg_getMaximalLength);
	dim3 B1(Block_reg_getMaximalLength,1,1);
	dim3 G1(Grid_reg_getMaximalLength,1,1);

	float *all_d;
    CUDA_SAFE_CALL(hipMalloc(&all_d, threadNumber*sizeof(float)));
	reg_getMaximalLength_kernel <<< G1, B1 >>> (all_d);
	CUDA_SAFE_CALL(hipDeviceSynchronize());
#ifndef NDEBUG
    printf("[NiftyReg CUDA DEBUG] reg_getMaximalLength kernel: %s - Grid size [%i %i %i] - Block size [%i %i %i]\n",
	       hipGetErrorString(hipGetLastError()),G1.x,G1.y,G1.z,B1.x,B1.y,B1.z);
#endif
    float *all_h;CUDA_SAFE_CALL(hipHostMalloc(&all_h, nodeNumber*sizeof(float)));
	CUDA_SAFE_CALL(hipMemcpy(all_h, all_d, threadNumber*sizeof(float),hipMemcpyDeviceToHost));
	CUDA_SAFE_CALL(hipFree(all_d));
	double maxDistance = 0.0f;
	for(int i=0; i<threadNumber; i++) maxDistance = all_h[i]>maxDistance?all_h[i]:maxDistance;
	CUDA_SAFE_CALL(hipHostFree((void *)all_h));

	return (float)maxDistance;
}

void reg_updateControlPointPosition_gpu(nifti_image *controlPointImage,
					float4 **controlPointImageArray_d,
					float4 **bestControlPointPosition_d,
					float4 **nodeNMIGradientArray_d,
					float currentLength)
{
	const int nodeNumber = controlPointImage->nx * controlPointImage->ny * controlPointImage->nz;
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_NodeNumber),&nodeNumber,sizeof(int)));
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_ScalingFactor),&currentLength,sizeof(float)));

	CUDA_SAFE_CALL(hipBindTexture(0, controlPointTexture, *bestControlPointPosition_d, nodeNumber*sizeof(float4)));
	CUDA_SAFE_CALL(hipBindTexture(0, gradientImageTexture, *nodeNMIGradientArray_d, nodeNumber*sizeof(float4)));

	const unsigned int Grid_reg_updateControlPointPosition = (unsigned int)ceil((float)nodeNumber/(float)Block_reg_updateControlPointPosition);
	dim3 B1(Block_reg_updateControlPointPosition,1,1);
	dim3 G1(Grid_reg_updateControlPointPosition,1,1);

	reg_updateControlPointPosition_kernel <<< G1, B1 >>> (*controlPointImageArray_d);
	CUDA_SAFE_CALL(hipDeviceSynchronize());
#ifndef NDEBUG
    printf("[NiftyReg CUDA DEBUG] reg_updateControlPointPosition kernel: %s - Grid size [%i %i %i] - Block size [%i %i %i]\n",
	       hipGetErrorString(hipGetLastError()),G1.x,G1.y,G1.z,B1.x,B1.y,B1.z);
#endif
}

void reg_gaussianSmoothing_gpu( nifti_image *image,
                                float4 **imageArray_d,
                                float sigma,
                                bool smoothXYZ[8])

{
    const int voxelNumber = image->nx * image->ny * image->nz;
    const int3 imageDim = make_int3(image->nx, image->ny, image->nz);

    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_ImageDim), &imageDim,sizeof(int3)));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_VoxelNumber), &voxelNumber,sizeof(int3)));

    bool axisToSmooth[8];
    if(smoothXYZ==NULL){
        for(int i=0; i<8; i++) axisToSmooth[i]=true;
    }
    else{
        for(int i=0; i<8; i++) axisToSmooth[i]=smoothXYZ[i];
    }

    for(int n=1; n<4; n++){
        if(axisToSmooth[n]==true){
            float currentSigma;
            if(sigma>0) currentSigma=sigma/image->pixdim[n];
            else currentSigma=fabs(sigma); // voxel based if negative value
            int radius=(int)ceil(currentSigma*3.0f);
            if(radius>0){
                int kernelSize = 1+radius*2;
                float *kernel_h;
                CUDA_SAFE_CALL(hipHostMalloc(&kernel_h, kernelSize*sizeof(float)));
                float kernelSum=0;
                for(int i=-radius; i<=radius; i++){
                    kernel_h[radius+i]=(float)(exp( -(i*i)/(2.0*currentSigma*currentSigma)) / (currentSigma*2.506628274631)); // 2.506... = sqrt(2*pi)
                    kernelSum += kernel_h[radius+i];
                }
                for(int i=0; i<kernelSize; i++)
                    kernel_h[i] /= kernelSum;
                float *kernel_d;
                CUDA_SAFE_CALL(hipMalloc(&kernel_d, kernelSize*sizeof(float)));
                CUDA_SAFE_CALL(hipMemcpy(kernel_d, kernel_h, kernelSize*sizeof(float), hipMemcpyHostToDevice));
                CUDA_SAFE_CALL(hipHostFree(kernel_h));

                float4 *smoothedImage;
                CUDA_SAFE_CALL(hipMalloc(&smoothedImage,voxelNumber*sizeof(float4)));

                CUDA_SAFE_CALL(hipBindTexture(0, convolutionKernelTexture, kernel_d, kernelSize*sizeof(float)));
                CUDA_SAFE_CALL(hipBindTexture(0, gradientImageTexture, *imageArray_d, voxelNumber*sizeof(float4)));
                unsigned int Grid_reg_ApplyConvolutionWindow;
                dim3 B,G;
                switch(n){
                    case 1:
                        Grid_reg_ApplyConvolutionWindow =
                            (unsigned int)ceil((float)voxelNumber/(float)Block_reg_ApplyConvolutionWindowAlongX);
                        B=dim3(Block_reg_ApplyConvolutionWindowAlongX,1,1);
                        G=dim3(Grid_reg_ApplyConvolutionWindow,1,1);
                        _reg_ApplyConvolutionWindowAlongX_kernel <<< G, B >>> (smoothedImage, kernelSize);
                        CUDA_SAFE_CALL(hipDeviceSynchronize());
#ifndef NDEBUG
                        printf("[NiftyReg CUDA DEBUG] reg_ApplyConvolutionWindowAlongX_kernel: %s - Grid size [%i %i %i] - Block size [%i %i %i]\n",
                            hipGetErrorString(hipGetLastError()),G.x,G.y,G.z,B.x,B.y,B.z);
#endif
                        break;
                    case 2:
                        Grid_reg_ApplyConvolutionWindow =
                            (unsigned int)ceil((float)voxelNumber/(float)Block_reg_ApplyConvolutionWindowAlongY);
                        B=dim3(Block_reg_ApplyConvolutionWindowAlongY,1,1);
                        G=dim3(Grid_reg_ApplyConvolutionWindow,1,1);
                        _reg_ApplyConvolutionWindowAlongY_kernel <<< G, B >>> (smoothedImage, kernelSize);
                        CUDA_SAFE_CALL(hipDeviceSynchronize());
#ifndef NDEBUG
                        printf("[NiftyReg CUDA DEBUG] reg_ApplyConvolutionWindowAlongY_kernel: %s - Grid size [%i %i %i] - Block size [%i %i %i]\n",
                            hipGetErrorString(hipGetLastError()),G.x,G.y,G.z,B.x,B.y,B.z);
#endif
                        break;
                    case 3:
                        Grid_reg_ApplyConvolutionWindow =
                            (unsigned int)ceil((float)voxelNumber/(float)Block_reg_ApplyConvolutionWindowAlongZ);
                        B=dim3(Block_reg_ApplyConvolutionWindowAlongZ,1,1);
                        G=dim3(Grid_reg_ApplyConvolutionWindow,1,1);
                        _reg_ApplyConvolutionWindowAlongZ_kernel <<< G, B >>> (smoothedImage, kernelSize);
                        CUDA_SAFE_CALL(hipDeviceSynchronize());
#ifndef NDEBUG
                        printf("[NiftyReg CUDA DEBUG] reg_ApplyConvolutionWindowAlongZ_kernel: %s - Grid size [%i %i %i] - Block size [%i %i %i]\n",
                            hipGetErrorString(hipGetLastError()),G.x,G.y,G.z,B.x,B.y,B.z);
#endif
                        break;
                }
                CUDA_SAFE_CALL(hipFree(kernel_d));
                CUDA_SAFE_CALL(hipMemcpy(*imageArray_d, smoothedImage, voxelNumber*sizeof(float4), hipMemcpyDeviceToDevice));
                CUDA_SAFE_CALL(hipFree(smoothedImage));
            }
        }
    }

}


void reg_smoothImageForCubicSpline_gpu( nifti_image *image,
                                        float4 **imageArray_d,
                                        int *smoothingRadius)
{
    const int voxelNumber = image->nx * image->ny * image->nz;
    const int3 imageDim = make_int3(image->nx, image->ny, image->nz);

    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_ImageDim), &imageDim,sizeof(int3)));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(c_VoxelNumber), &voxelNumber,sizeof(int3)));

    for(int n=0; n<3; n++){
        if(smoothingRadius[n]>0){
            int kernelSize = 1+smoothingRadius[n]*2;
            float *kernel_h;
            CUDA_SAFE_CALL(hipHostMalloc(&kernel_h, kernelSize*sizeof(float)));
//             float kernelSum=0;
            for(int i=-smoothingRadius[n]; i<=smoothingRadius[n]; i++){
                float coeff = fabs(2.0f*(float)(i)/smoothingRadius[n]);
                if(coeff<1.0f)  kernel_h[smoothingRadius[n]+i] = 2.0f/3.0f - coeff*coeff + 0.5f*coeff*coeff*coeff;
                else        kernel_h[smoothingRadius[n]+i] = -(coeff-2.0f)*(coeff-2.0f)*(coeff-2.0f)/6.0f;
//                 kernelSum += kernel_h[smoothingRadius[n]+i];
            }
//             for(int i=0; i<kernelSize; i++) kernel_h[i] /= kernelSum;
            float *kernel_d;
            CUDA_SAFE_CALL(hipMalloc(&kernel_d, kernelSize*sizeof(float)));
            CUDA_SAFE_CALL(hipMemcpy(kernel_d, kernel_h, kernelSize*sizeof(float), hipMemcpyHostToDevice));
            CUDA_SAFE_CALL(hipHostFree(kernel_h));
            CUDA_SAFE_CALL(hipBindTexture(0, convolutionKernelTexture, kernel_d, kernelSize*sizeof(float)));

            float4 *smoothedImage_d;
            CUDA_SAFE_CALL(hipMalloc(&smoothedImage_d,voxelNumber*sizeof(float4)));

            CUDA_SAFE_CALL(hipBindTexture(0, gradientImageTexture, *imageArray_d, voxelNumber*sizeof(float4)));

            unsigned int Grid_reg_ApplyConvolutionWindow;
            dim3 B,G;
            switch(n){
                case 0:
                    Grid_reg_ApplyConvolutionWindow =
                        (unsigned int)ceil((float)voxelNumber/(float)Block_reg_ApplyConvolutionWindowAlongX);
                    B=dim3(Block_reg_ApplyConvolutionWindowAlongX,1,1);
                    G=dim3(Grid_reg_ApplyConvolutionWindow,1,1);
                    _reg_ApplyConvolutionWindowAlongX_kernel <<< G, B >>> (smoothedImage_d, kernelSize);
                    CUDA_SAFE_CALL(hipDeviceSynchronize());
#ifndef NDEBUG
                    printf("[NiftyReg CUDA DEBUG] reg_ApplyConvolutionWindowAlongX_kernel: %s - Grid size [%i %i %i] - Block size [%i %i %i]\n",
                        hipGetErrorString(hipGetLastError()),G.x,G.y,G.z,B.x,B.y,B.z);
#endif
                    break;
                case 1:
                    Grid_reg_ApplyConvolutionWindow =
                        (unsigned int)ceil((float)voxelNumber/(float)Block_reg_ApplyConvolutionWindowAlongY);
                    B=dim3(Block_reg_ApplyConvolutionWindowAlongY,1,1);
                    G=dim3(Grid_reg_ApplyConvolutionWindow,1,1);
                    _reg_ApplyConvolutionWindowAlongY_kernel <<< G, B >>> (smoothedImage_d, kernelSize);
                    CUDA_SAFE_CALL(hipDeviceSynchronize());
#ifndef NDEBUG
                    printf("[NiftyReg CUDA DEBUG] reg_ApplyConvolutionWindowAlongY_kernel: %s - Grid size [%i %i %i] - Block size [%i %i %i]\n",
                        hipGetErrorString(hipGetLastError()),G.x,G.y,G.z,B.x,B.y,B.z);
#endif
                    break;
                case 2:
                    Grid_reg_ApplyConvolutionWindow =
                        (unsigned int)ceil((float)voxelNumber/(float)Block_reg_ApplyConvolutionWindowAlongZ);
                    B=dim3(Block_reg_ApplyConvolutionWindowAlongZ,1,1);
                    G=dim3(Grid_reg_ApplyConvolutionWindow,1,1);
                    _reg_ApplyConvolutionWindowAlongZ_kernel <<< G, B >>> (smoothedImage_d, kernelSize);
                    CUDA_SAFE_CALL(hipDeviceSynchronize());
#ifndef NDEBUG
                    printf("[NiftyReg CUDA DEBUG] reg_ApplyConvolutionWindowAlongZ_kernel: %s - Grid size [%i %i %i] - Block size [%i %i %i]\n",
                        hipGetErrorString(hipGetLastError()),G.x,G.y,G.z,B.x,B.y,B.z);
#endif
                    break;
            }
            CUDA_SAFE_CALL(hipFree(kernel_d));
            CUDA_SAFE_CALL(hipMemcpy(*imageArray_d, smoothedImage_d, voxelNumber*sizeof(float4), hipMemcpyDeviceToDevice));
            CUDA_SAFE_CALL(hipFree(smoothedImage_d));
        }
    }
}

#endif

